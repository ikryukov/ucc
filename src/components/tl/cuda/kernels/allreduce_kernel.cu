#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "utils/arch/cuda_def.h"
#include "../tl_cuda.h"
#include "ucc/api/ucc.h"

#ifdef __cplusplus
}
#endif

#include "nvls.cuh"

// vectorized allreduce kernel for 32-bit lanes
template <typename NvlsOps>
__global__ void __launch_bounds__(UCC_TL_CUDA_MAX_NVLS_THREADS)
allreduce_kernel_vec32(uint32_t *base_u32, size_t count_u32, uint32_t rank,
                       uint32_t tsize)
{
    size_t chunk_start = ((int64_t)count_u32 * (int64_t)rank) / (int64_t)tsize;
    size_t chunk_end   = ((int64_t)count_u32 * (int64_t)(rank + 1)) / (int64_t)tsize;

    size_t thread_offset = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    size_t stride        = blockDim.x * gridDim.x * 4;

    for (size_t idx = chunk_start + thread_offset; idx < chunk_end; idx += stride) {
        uint4 val;
        NvlsOps::ld(val, base_u32 + idx);
        NvlsOps::st(val, base_u32 + idx);
    }
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t post_allreduce_kernel(hipStream_t stream, uint32_t sm_count,
                                   uint32_t threads, hipDeviceptr_t src_addr,
                                   size_t src_size_bytes, uint32_t rank,
                                   uint32_t tsize, ucc_datatype_t datatype)
{
    assert(sm_count > 0 && sm_count <= UCC_TL_CUDA_MAX_NVLS_SM_COUNT);
    assert(threads > 0 && threads <= UCC_TL_CUDA_MAX_NVLS_THREADS);
    uint32_t *base_u32   = reinterpret_cast<uint32_t *>(src_addr);
    size_t    count_u32  = src_size_bytes / sizeof(uint32_t);

    switch (datatype) {
    case UCC_DT_FLOAT32:
        assert(((uintptr_t)(src_addr) % 8) == 0);
        allreduce_kernel_vec32<NvlsFp32Ops><<<sm_count, threads, 0, stream>>>(
            base_u32, count_u32, rank, tsize);
        break;
    case UCC_DT_BFLOAT16:
        assert(((uintptr_t)(src_addr) % 8) == 0);
        allreduce_kernel_vec32<NvlsBf16Ops><<<sm_count, threads, 0, stream>>>(
            base_u32, count_u32, rank, tsize);
        break;
    default:
        return UCC_ERR_NOT_SUPPORTED;
    }
    CUDA_CHECK(hipGetLastError());

    return UCC_OK;
}

#ifdef __cplusplus
}
#endif

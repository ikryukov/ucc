#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2025, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See file LICENSE for terms.
 */

#ifdef __cplusplus
extern "C" {
#endif

#include "utils/arch/cuda_def.h"
#include "../tl_cuda.h"
#include "ucc/api/ucc.h"

#include "nvls.cuh"

#ifdef __cplusplus
}
#endif

#include <hip/hip_bf16.h>

__global__ void __launch_bounds__(UCC_TL_CUDA_MAX_NVLS_THREADS)
    allreduce_kernel_fp32(float *src_addr, size_t src_count, uint32_t rank,
                          uint32_t tsize)
{
    size_t chunk_start = ((int64_t)src_count * (int64_t)rank) / (int64_t)tsize;
    size_t chunk_end =
        ((int64_t)src_count * (int64_t)(rank + 1)) / (int64_t)tsize;

    size_t thread_offset = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    size_t stride        = blockDim.x * gridDim.x * 4;

    for (size_t idx = chunk_start + thread_offset; idx < chunk_end;
         idx += stride) {
        uint4 val;
        MULTIMEM_LD(val, src_addr + idx);
        MULTIMEM_ST(val, src_addr + idx);
    }

    return;
}

__global__ void __launch_bounds__(UCC_TL_CUDA_MAX_NVLS_THREADS)
    allreduce_kernel_bfloat16(float *src_addr, size_t src_count, uint32_t rank,
                              uint32_t tsize)
{
    size_t chunk_start = ((int64_t)src_count * (int64_t)rank) / (int64_t)tsize;
    size_t chunk_end =
        ((int64_t)src_count * (int64_t)(rank + 1)) / (int64_t)tsize;

    size_t thread_offset = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    size_t stride        = blockDim.x * gridDim.x * 4;

    for (size_t idx = chunk_start + thread_offset; idx < chunk_end;
         idx += stride) {
        uint4 val;
        MULTIMEM_LD_BF16(val, src_addr + idx);
        MULTIMEM_ST_BF16(val, src_addr + idx);
    }

    return;
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t post_allreduce_kernel(hipStream_t stream, uint32_t sm_count,
                                   uint32_t threads, hipDeviceptr_t src_addr,
                                   size_t src_size_bytes, uint32_t rank,
                                   uint32_t tsize, ucc_datatype_t datatype)
{
    assert(sm_count > 0 && sm_count <= UCC_TL_CUDA_MAX_NVLS_SM_COUNT);
    assert(threads > 0 && threads <= UCC_TL_CUDA_MAX_NVLS_THREADS);
    switch (datatype) {
    case UCC_DT_FLOAT32:
        allreduce_kernel_fp32<<<sm_count, threads, 0, stream>>>(
            (float *)src_addr, src_size_bytes / sizeof(float), rank, tsize);
        break;
    case UCC_DT_BFLOAT16:
        assert(((uintptr_t)(src_addr) % 8) == 0);
        allreduce_kernel_bfloat16<<<sm_count, threads, 0, stream>>>(
            (float *)src_addr, src_size_bytes / sizeof(float), rank, tsize);
        break;
    default:
        return UCC_ERR_NOT_SUPPORTED;
    }
    CUDA_CHECK(hipGetLastError());

    return UCC_OK;
}

#ifdef __cplusplus
}
#endif
